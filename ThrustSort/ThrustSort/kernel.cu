#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <cstdio>

struct MyItem
{
    int a;
    int b;

    __host__ __device__ bool operator < (const MyItem& rhs) const
    {
        return (a + b) < (rhs.a + rhs.b);
    }

    __host__ void PrintItemValue(void)
    {
        printf("(%d + %d = %d)\n", a, b, a + b);
    }
};

static void ItemSortTest(void)
{
    constexpr int elemCount = 16;

    MyItem items[elemCount] = {
        MyItem{ 0, 1 }, MyItem{ 5, 6 }, MyItem{ 2, 3 }, MyItem{ 9, 9 },
        MyItem{ 8, 8 }, MyItem{ 7, 7 }, MyItem{ 6, 7 }, MyItem{ 6, 6 },
        MyItem{ 5, 5 }, MyItem{ 4, 4 }, MyItem{ 3, 3 }, MyItem{ 4, 5 },
        MyItem{ 2, 2 }, MyItem{ 0, 0 }, MyItem{ 1, 2 }, MyItem{ 3, 6 }
    };

    MyItem* devMem = nullptr;
    constexpr auto bufferSize = elemCount * sizeof(*devMem);

    do
    {
        auto cudaStatus = hipMalloc(&devMem, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devMem, items, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        try
        {
            thrust::device_ptr<MyItem> sortItems(devMem);
            thrust::sort(sortItems, sortItems + elemCount, thrust::less<MyItem>());
        }
        catch (const thrust::system::system_error& sysErr)
        {
            printf("thrust system error: %s\n", sysErr.what());
            break;
        }
        catch (const thrust::system::detail::bad_alloc& allocErr)
        {
            printf("thrust bad alloc: %s\n", allocErr.what());
            break;
        }

        cudaStatus = hipMemcpy(items, devMem, bufferSize, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        for (int i = 0; i < elemCount; i++) {
            items[i].PrintItemValue();
        }

    } while (false);

    if (devMem != nullptr) {
        hipFree(devMem);
    }
}

static void KeySortTest(void)
{
    //  0123456789abcdef
    // "Hi, CUDA Thrust!"
    int chars[] = {
        'A', 'C', 'D', 'H', 'h', 'i', 'r', ' ',
        's', 'T', 't', 'U', 'u', '!', ',', ' '
    };
    const int keys[] = {
        7, 4, 6, 0, 10, 1, 11, 3,
        13, 9, 14, 5, 12, 15, 2, 8
    };

    constexpr auto bufferSize = sizeof(keys);
    constexpr int elemCount = int(bufferSize / sizeof(keys[0]));

    int* devChars = nullptr;
    int* devKeys = nullptr;

    do
    {
        auto cudaStatus = hipMalloc(&devChars, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMalloc(&devKeys, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devChars, chars, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devKeys, keys, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        try
        {
            thrust::device_ptr<int> sortItems(devChars);
            thrust::device_ptr<int> sortKeys(devKeys);
            thrust::sort_by_key(sortKeys, sortKeys + elemCount, sortItems, thrust::less<int>());
        }
        catch (const thrust::system::system_error& sysErr)
        {
            printf("thrust system error: %s\n", sysErr.what());
            break;
        }
        catch (const thrust::system::detail::bad_alloc& allocErr)
        {
            printf("thrust bad alloc: %s\n", allocErr.what());
            break;
        }

        cudaStatus = hipMemcpy(chars, devChars, bufferSize, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        printf("The string is: ");
        for (int i = 0; i < elemCount; i++) {
            printf("%c", chars[i]);
        }
        puts("");

    } while (false);

    if (devChars != nullptr) {
        hipFree(devChars);
    }
    if (devKeys != nullptr) {
        hipFree(devKeys);
    }
}

int main(void)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        puts("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 0;
    }

    ItemSortTest();

    puts("\n================\n");

    KeySortTest();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        puts("hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

